#include <iostream>
#include "sparsebase/sparse_format.h"
#include "sparsebase/cuda/format.cuh"
#include "sparsebase/sparse_exception.h"

namespace sparsebase {

namespace context {
  HIPContext::HIPContext(int did): device_id(did){
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_id >= device_count){
      throw utils::CUDADeviceException(device_count, device_id);
    }
  }
  bool HIPContext::IsEquivalent(Context * rhs) const {
    if (dynamic_cast<HIPContext*>(rhs)!= nullptr){
      if (dynamic_cast<HIPContext*>(rhs)->device_id== this->device_id)
        return true;
    } 
    return false;
  }
}

namespace format {

template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::CUDACSR(CUDACSR<IDType, NNZType, ValueType> &&rhs)
    : col_(std::move(rhs.col_)), row_ptr_(std::move(rhs.row_ptr_)),
      vals_(std::move(rhs.vals_)) {
  this->nnz_ = rhs.get_num_nnz();
  this->order_ = 2;
  this->dimension_ = rhs.dimension_;
  rhs.col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      nullptr, BlankDeleter<IDType>());
  rhs.row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      nullptr, BlankDeleter<NNZType>());
  rhs.vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      nullptr, BlankDeleter<ValueType>());
  this->context_ = std::unique_ptr<sparsebase::context::Context>(new sparsebase::context::HIPContext(rhs.get_cuda_context()->device_id));
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType> &CUDACSR<IDType, NNZType, ValueType>::operator=(
    const CUDACSR<IDType, NNZType, ValueType> &rhs) {
  this->nnz_ = rhs.nnz_;
  this->order_ = 2;
  this->dimension_ = rhs.dimension_;
  IDType * col;
  NNZType * row_ptr;
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(this->get_cuda_context());
  hipSetDevice(gpu_context->device_id);
  hipMalloc(&col, rhs.get_num_nnz()*sizeof(IDType));
  hipMemcpy(col, rhs.get_col(), rhs.get_num_nnz()*sizeof(IDType), hipMemcpyDeviceToDevice);
  hipMalloc(&row_ptr, (rhs.get_dimensions()[0]+1)*sizeof(NNZType));
  hipMemcpy(row_ptr, rhs.get_row_ptr(), (rhs.get_dimensions()[0]+1)*sizeof(NNZType), hipMemcpyDeviceToDevice);
  ValueType *vals = nullptr;
  if (rhs.get_vals() != nullptr) {
    hipMalloc(&vals, rhs.get_num_nnz()*sizeof(ValueType));
    hipMemcpy(vals, rhs.get_vals(), rhs.get_num_nnz()*sizeof(ValueType), hipMemcpyDeviceToDevice);
  }
  this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      col, CUDADeleter<IDType>());
  this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      row_ptr, CUDADeleter<NNZType>());
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, CUDADeleter<ValueType>());
  return *this;
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::CUDACSR(const CUDACSR<IDType, NNZType, ValueType> &rhs)
    : col_(nullptr, BlankDeleter<IDType>()),
      row_ptr_(nullptr, BlankDeleter<NNZType>()),
      vals_(nullptr, BlankDeleter<ValueType>()) {
  this->nnz_ = rhs.nnz_;
  this->order_ = 2;
  this->dimension_ = rhs.dimension_;
  IDType * col;
  NNZType * row_ptr;
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(this->get_context());
  hipSetDevice(gpu_context->device_id);
  hipMalloc(&col, rhs.get_num_nnz()*sizeof(IDType));
  hipMemcpy(col, rhs.get_col(), rhs.get_num_nnz()*sizeof(IDType), hipMemcpyDeviceToDevice);
  hipMalloc(&row_ptr, rhs.get_dimensions()[0]*sizeof(NNZType));
  hipMemcpy(row_ptr, rhs.get_row_ptr(), (rhs.get_dimensions()[0]+1)*sizeof(NNZType), hipMemcpyDeviceToDevice);
  ValueType *vals = nullptr;
  if (rhs.get_vals() != nullptr) {
    hipMalloc(&vals, rhs.get_num_nnz()*sizeof(ValueType));
    hipMemcpy(vals, rhs.get_vals(), rhs.get_num_nnz()*sizeof(ValueType), hipMemcpyDeviceToDevice);
  }
  this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      col, CUDADeleter<IDType>());
  this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      row_ptr, CUDADeleter<NNZType>());
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, CUDADeleter<ValueType>());
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::CUDACSR(IDType n, IDType m, NNZType nnz, NNZType *row_ptr,
                                     IDType *col, ValueType *vals, context::HIPContext context,
                                     Ownership own)
    : row_ptr_(row_ptr, BlankDeleter<NNZType>()),
      col_(col, BlankDeleter<IDType>()),
      vals_(vals, BlankDeleter<ValueType>()) {
  this->order_ = 2;
  this->dimension_ = {(DimensionType)n, (DimensionType)m};
  this->nnz_ = nnz;
  if (own == kOwned) {
    this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
        row_ptr, CUDADeleter<NNZType>());
    this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
        col, CUDADeleter<IDType>());
    this->vals_ =
        std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
            vals, CUDADeleter<ValueType>());
  }
  this->context_ = std::unique_ptr<sparsebase::context::Context>(new sparsebase::context::HIPContext(context));
}

template <typename IDType, typename NNZType, typename ValueType>
context::HIPContext* CUDACSR<IDType, NNZType, ValueType>::get_cuda_context() const{
  return static_cast<context::HIPContext*>(this->get_context());
}
template <typename IDType, typename NNZType, typename ValueType>
Format *CUDACSR<IDType, NNZType, ValueType>::clone() const {
  return new CUDACSR(*this);
}
template <typename IDType, typename NNZType, typename ValueType>
IDType *CUDACSR<IDType, NNZType, ValueType>::get_col() const {
  return col_.get();
}
template <typename IDType, typename NNZType, typename ValueType>
NNZType *CUDACSR<IDType, NNZType, ValueType>::get_row_ptr() const {
  return row_ptr_.get();
}
template <typename IDType, typename NNZType, typename ValueType>
ValueType *CUDACSR<IDType, NNZType, ValueType>::get_vals() const {
  return vals_.get();
}
template <typename IDType, typename NNZType, typename ValueType>
IDType *CUDACSR<IDType, NNZType, ValueType>::release_col() {
  auto col = col_.release();
  this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      col, BlankDeleter<IDType>());
  return col;
}
template <typename IDType, typename NNZType, typename ValueType>
NNZType *CUDACSR<IDType, NNZType, ValueType>::release_row_ptr() {
  auto row_ptr = row_ptr_.release();
  this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      row_ptr, BlankDeleter<NNZType>());
  return row_ptr;
}
template <typename IDType, typename NNZType, typename ValueType>
ValueType *CUDACSR<IDType, NNZType, ValueType>::release_vals() {
  auto vals = vals_.release();
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, BlankDeleter<ValueType>());
  return vals;
}

template <typename IDType, typename NNZType, typename ValueType>
void CUDACSR<IDType, NNZType, ValueType>::set_col(IDType *col, context::HIPContext context, Ownership own) {
  if (own == kOwned) {
    this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
        col, CUDADeleter<IDType>());
  } else {
    this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
        col, BlankDeleter<IDType>());
  }
}

template <typename IDType, typename NNZType, typename ValueType>
void CUDACSR<IDType, NNZType, ValueType>::set_row_ptr(NNZType *row_ptr, context::HIPContext context,
                                                  Ownership own) {
  if (own == kOwned) {
    this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
        row_ptr, CUDADeleter<NNZType>());
  } else {
    this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
        row_ptr, BlankDeleter<NNZType>());
  }
}

template <typename IDType, typename NNZType, typename ValueType>
void CUDACSR<IDType, NNZType, ValueType>::set_vals(ValueType *vals, context::HIPContext context, Ownership own) {
  if (own == kOwned) {
    this->vals_ =
        std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
            vals, CUDADeleter<ValueType>());
  } else {
    this->vals_ =
        std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
            vals, BlankDeleter<ValueType>());
  }
}

template <typename IDType, typename NNZType, typename ValueType>
bool CUDACSR<IDType, NNZType, ValueType>::RowPtrIsOwned() {
  return (this->row_ptr_.get_deleter().target_type() !=
          typeid(BlankDeleter<NNZType>));
}

template <typename IDType, typename NNZType, typename ValueType>
bool CUDACSR<IDType, NNZType, ValueType>::ColIsOwned() {
  return (this->col_.get_deleter().target_type() !=
          typeid(BlankDeleter<IDType>));
}

template <typename IDType, typename NNZType, typename ValueType>
bool CUDACSR<IDType, NNZType, ValueType>::ValsIsOwned() {
  return (this->vals_.get_deleter().target_type() !=
          typeid(BlankDeleter<ValueType>));
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::~CUDACSR() {}

template <typename ValueType>
CUDAArray<ValueType>::CUDAArray(CUDAArray<ValueType> &&rhs):
      vals_(std::move(rhs.vals_)) {
  this->nnz_ = rhs.get_num_nnz();
  this->order_ = 1;
  this->dimension_ = rhs.dimension_;
  rhs.vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      nullptr, BlankDeleter<ValueType>());
  this->context_ = std::unique_ptr<sparsebase::context::Context>(new sparsebase::context::CPUContext);
}
template <typename ValueType>
CUDAArray<ValueType> &CUDAArray<ValueType>::operator=(
    const CUDAArray<ValueType> &rhs) {
  this->nnz_ = rhs.nnz_;
  this->order_ = 1;
  this->dimension_ = rhs.dimension_;
  ValueType *vals = nullptr;
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(this->get_context());
  if (rhs.get_vals() != nullptr) {
    hipSetDevice(gpu_context->device_id);
    hipMalloc(&vals, rhs.get_num_nnz()*sizeof(ValueType));
    hipMemcpy(vals, rhs.get_vals(), rhs.get_num_nnz()*sizeof(ValueType), hipMemcpyDeviceToDevice);
    vals = new ValueType[rhs.get_num_nnz()];
    std::copy(rhs.get_vals(), rhs.get_vals() + rhs.get_num_nnz(), vals);
  }
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, CUDADeleter<ValueType>());
  return *this;
}
template <typename ValueType>
CUDAArray<ValueType>::CUDAArray(const CUDAArray<ValueType> &rhs)
    : vals_(nullptr, BlankDeleter<ValueType>()) {
  this->nnz_ = rhs.nnz_;
  this->order_ = 1;
  this->dimension_ = rhs.dimension_;
  ValueType *vals = nullptr;
  if (rhs.get_vals() != nullptr) {
    vals = new ValueType[rhs.get_num_nnz()];
    std::copy(rhs.get_vals(), rhs.get_vals() + rhs.get_num_nnz(), vals);
  }
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, Deleter<ValueType>());
  this->context_ = std::unique_ptr<sparsebase::context::Context>(new sparsebase::context::CPUContext);
}
template <typename ValueType>
CUDAArray<ValueType>::CUDAArray(DimensionType nnz, ValueType* vals, context::HIPContext context, Ownership own)
    :  vals_(vals, BlankDeleter<ValueType>()) {
  this->order_ = 1;
  this->dimension_ = {(DimensionType)nnz};
  this->nnz_ = nnz;
  if (own == kOwned) {
    this->vals_ =
        std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
            vals, CUDADeleter<ValueType>());
  }
  this->context_ = std::unique_ptr<sparsebase::context::Context>(new sparsebase::context::HIPContext(context));
}

template <typename ValueType>
Format *CUDAArray<ValueType>::clone() const {
  return new CUDAArray(*this);
}
template <typename ValueType>
ValueType *CUDAArray<ValueType>::get_vals() const {
  return vals_.get();
}
template <typename ValueType>
ValueType *CUDAArray<ValueType>::release_vals() {
  auto vals = vals_.release();
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, BlankDeleter<ValueType>());
  return vals;
}

template <typename ValueType>
void CUDAArray<ValueType>::set_vals(ValueType *vals, Ownership own) {
  if (own == kOwned) {
    this->vals_ =
        std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
            vals, Deleter<ValueType>());
  } else {
    this->vals_ =
        std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
            vals, BlankDeleter<ValueType>());
  }
}

template <typename ValueType>
bool CUDAArray<ValueType>::ValsIsOwned() {
  return (this->vals_.get_deleter().target_type() !=
          typeid(BlankDeleter<ValueType>));
}
template <typename ValueType>
CUDAArray<ValueType>::~CUDAArray() {}
// format.inc

#if !defined(_HEADER_ONLY)
#include "init/cuda/format.inc"
#endif
};
};