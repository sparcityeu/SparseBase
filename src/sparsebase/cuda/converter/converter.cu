#include "sparsebase/sparse_format.h"
#include "sparsebase/sparse_converter.h"
#include "sparsebase/cuda/format.cuh"

using namespace sparsebase;
using namespace sparsebase::format;
template <typename ValueType>
Format *
CUDAArrayArrayConditionalFunction(Format *source, context::Context*context) {
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(source->get_context());
  auto cuda_array = source->As<CUDAArray<ValueType>>();
  hipSetDevice(gpu_context->device_id);
  ValueType * vals = nullptr; 
  if (cuda_array->get_vals()!=nullptr){
    vals = new ValueType[cuda_array->get_num_nnz()];
    hipMemcpy(vals, cuda_array->get_vals(), cuda_array->get_num_nnz() * sizeof(ValueType),
               hipMemcpyDeviceToHost);
  }
  return new Array<ValueType>(cuda_array->get_num_nnz(), vals);
}
template <typename ValueType>
Format *
ArrayCUDAArrayConditionalFunction(Format *source, context::Context*context) {
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(context);
  auto array = source->As<Array<ValueType>>();
  hipSetDevice(gpu_context->device_id);
  ValueType * vals = nullptr; 
  if (array->get_vals()!=nullptr){
    hipMalloc(&vals, array->get_num_nnz() * sizeof(ValueType));
    hipMemcpy(vals, array->get_vals(), array->get_num_nnz() * sizeof(ValueType),
               hipMemcpyHostToDevice);
  }
  return new CUDAArray<ValueType>(array->get_num_nnz(), vals, *gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
Format *
CsrCUDACsrConditionalFunction(Format *source, context::Context*context) {
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(context);
  auto csr = source->As<CSR<IDType, NNZType, ValueType>>();
  hipSetDevice(gpu_context->device_id);
  NNZType * row_ptr;
  IDType * col;
  ValueType * vals = nullptr; 
  hipMalloc(&row_ptr, (1+csr->get_dimensions()[0])*sizeof(NNZType));
  hipMemcpy(row_ptr, csr->get_row_ptr(), (1+csr->get_dimensions()[0])*sizeof(NNZType), hipMemcpyHostToDevice);
  hipMalloc(&col, csr->get_num_nnz()*sizeof(IDType));
  hipMemcpy(col, csr->get_col(), csr->get_num_nnz()*sizeof(IDType), hipMemcpyHostToDevice);
  if (csr->get_vals()!=nullptr){
    hipMalloc(&vals, csr->get_num_nnz() * sizeof(ValueType));
    hipMemcpy(vals, csr->get_vals(), csr->get_num_nnz() * sizeof(ValueType),
               hipMemcpyHostToDevice);
  }
  return new CUDACSR<IDType, NNZType, ValueType>(csr->get_dimensions()[0], csr->get_dimensions()[0], csr->get_num_nnz(), row_ptr, col, vals, *gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
Format *
CUDACsrCUDACsrConditionalFunction(Format *source, context::Context*context) {
  context::HIPContext* dest_gpu_context = static_cast<context::HIPContext*>(context);
  auto cuda_csr = source->As<CUDACSR<IDType, NNZType, ValueType>>();
  context::HIPContext* source_gpu_context = static_cast<context::HIPContext*>(cuda_csr->get_context());
  hipSetDevice(dest_gpu_context->device_id);
  hipDeviceEnablePeerAccess(source_gpu_context->device_id,0);
  NNZType * row_ptr;
  IDType * col;
  ValueType * vals = nullptr; 
  hipMalloc(&row_ptr, cuda_csr->get_dimensions()[0]*sizeof(NNZType));
  hipMemcpy(row_ptr, cuda_csr->get_row_ptr(), (1+cuda_csr->get_dimensions()[0])*sizeof(NNZType), hipMemcpyDeviceToDevice);
  hipMalloc(&col, cuda_csr->get_num_nnz()*sizeof(IDType));
  hipMemcpy(col, cuda_csr->get_col(), cuda_csr->get_num_nnz()*sizeof(IDType), hipMemcpyDeviceToDevice);
  if (cuda_csr->get_vals()!=nullptr){
    hipMalloc(&vals, cuda_csr->get_num_nnz() * sizeof(ValueType));
    hipMemcpy(vals, cuda_csr->get_vals(), cuda_csr->get_num_nnz() * sizeof(ValueType),
               hipMemcpyDeviceToDevice);
  }
  return new CUDACSR<IDType, NNZType, ValueType>(cuda_csr->get_dimensions()[0], cuda_csr->get_dimensions()[0], cuda_csr->get_num_nnz(), row_ptr, col, vals, *dest_gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
Format *
CUDACsrCsrConditionalFunction(Format *source, context::Context*context) {
  context::HIPContext* gpu_context = static_cast<context::HIPContext*>(source->get_context());
  auto cuda_csr = source->As<CUDACSR<IDType, NNZType, ValueType>>();
  hipSetDevice(gpu_context->device_id);
  int n = cuda_csr->get_dimensions()[0];
  int nnz = cuda_csr->get_num_nnz();
  NNZType * row_ptr = new NNZType[n+1];
  IDType * col = new IDType[nnz];
  ValueType * vals = nullptr; 
  hipMemcpy(row_ptr, cuda_csr->get_row_ptr(), (n+1)*sizeof(NNZType), hipMemcpyDeviceToHost);
  hipMemcpy(col, cuda_csr->get_col(), nnz*sizeof(IDType), hipMemcpyDeviceToHost);
  if (cuda_csr->get_vals()!=nullptr){
    vals = new ValueType[nnz];
    hipMemcpy(vals, cuda_csr->get_vals(), nnz * sizeof(ValueType),
               hipMemcpyDeviceToHost);
  }
  return new CSR<IDType, NNZType, ValueType>(n, n, row_ptr, col, vals);
}

bool CUDAPeerToPeer(context::Context* from, context::Context* to){
  auto from_gpu = static_cast<context::HIPContext*>(from);
  auto to_gpu = static_cast<context::HIPContext*>(to);
  int can_access;
  hipDeviceCanAccessPeer(&can_access, from_gpu->device_id, to_gpu->device_id);
  return can_access;
}
#if !defined(_HEADER_ONLY)
#include "init/external/cuda/converter.inc"
#endif