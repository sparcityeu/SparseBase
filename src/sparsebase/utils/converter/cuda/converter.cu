#include "sparsebase/context/context.h"
#include "sparsebase/context/cuda/context.cuh"
#include "sparsebase/format/cuda/format.cuh"
#include "sparsebase/format/format.h"
#include "sparsebase/utils/converter/converter.h"
#include "sparsebase/utils/converter/cuda/converter.cuh"
#include "sparsebase/format/cuda/format.cuh"

namespace sparsebase {
namespace utils {
namespace converter {
namespace cuda {

template <typename ValueType>
format::Format *CUDAArrayArrayConditionalFunction(format::Format *source,
                                          context::Context *context) {
  context::cuda::HIPContext *gpu_context =
      static_cast<context::cuda::HIPContext *>(source->get_context());
  auto cuda_array = source->AsAbsolute<format::cuda::CUDAArray<ValueType>>();
  hipSetDevice(gpu_context->device_id);
  ValueType *vals = nullptr;
  if (cuda_array->get_vals() != nullptr) {
    vals = new ValueType[cuda_array->get_num_nnz()];
    hipMemcpy(vals, cuda_array->get_vals(),
               cuda_array->get_num_nnz() * sizeof(ValueType),
               hipMemcpyDeviceToHost);
  }
  return new format::Array<ValueType>(cuda_array->get_num_nnz(), vals);
}
template <typename ValueType>
format::Format *ArrayCUDAArrayConditionalFunction(format::Format *source,
                                          context::Context *context) {
  context::cuda::HIPContext *gpu_context =
      static_cast<context::cuda::HIPContext *>(context);
  auto array = source->AsAbsolute<format::Array<ValueType>>();
  hipSetDevice(gpu_context->device_id);
  ValueType *vals = nullptr;
  if (array->get_vals() != nullptr) {
    hipMalloc(&vals, array->get_num_nnz() * sizeof(ValueType));
    hipMemcpy(vals, array->get_vals(),
               array->get_num_nnz() * sizeof(ValueType),
               hipMemcpyHostToDevice);
  }
  return new format::cuda::CUDAArray<ValueType>(array->get_num_nnz(), vals,
                                                *gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
format::Format *CsrCUDACsrConditionalFunction(format::Format *source,
                                      context::Context *context) {
  context::cuda::HIPContext *gpu_context =
      static_cast<context::cuda::HIPContext *>(context);
  auto csr = source->AsAbsolute<format::CSR<IDType, NNZType, ValueType>>();
  hipSetDevice(gpu_context->device_id);
  NNZType *row_ptr;
  IDType *col;
  ValueType *vals = nullptr;
  hipMalloc(&row_ptr, (1 + csr->get_dimensions()[0]) * sizeof(NNZType));
  hipMemcpy(row_ptr, csr->get_row_ptr(),
             (1 + csr->get_dimensions()[0]) * sizeof(NNZType),
             hipMemcpyHostToDevice);
  hipMalloc(&col, csr->get_num_nnz() * sizeof(IDType));
  hipMemcpy(col, csr->get_col(), csr->get_num_nnz() * sizeof(IDType),
             hipMemcpyHostToDevice);
  if (csr->get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      hipMalloc(&vals, csr->get_num_nnz() * sizeof(ValueType));
      hipMemcpy(vals, csr->get_vals(), csr->get_num_nnz() * sizeof(ValueType),
                 hipMemcpyHostToDevice);
    }
  }
  return new format::cuda::CUDACSR<IDType, NNZType, ValueType>(
      csr->get_dimensions()[0], csr->get_dimensions()[0], csr->get_num_nnz(),
      row_ptr, col, vals, *gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
format::Format *CUDACsrCUDACsrConditionalFunction(format::Format *source,
                                          context::Context *context) {
  context::cuda::HIPContext *dest_gpu_context =
      static_cast<context::cuda::HIPContext *>(context);
  auto cuda_csr =
      source->AsAbsolute<format::cuda::CUDACSR<IDType, NNZType, ValueType>>();
  context::cuda::HIPContext *source_gpu_context =
      static_cast<context::cuda::HIPContext *>(cuda_csr->get_context());
  hipSetDevice(dest_gpu_context->device_id);
  hipDeviceEnablePeerAccess(source_gpu_context->device_id, 0);
  NNZType *row_ptr;
  IDType *col;
  ValueType *vals = nullptr;
  hipMalloc(&row_ptr, cuda_csr->get_dimensions()[0] * sizeof(NNZType));
  hipMemcpy(row_ptr, cuda_csr->get_row_ptr(),
             (1 + cuda_csr->get_dimensions()[0]) * sizeof(NNZType),
             hipMemcpyDeviceToDevice);
  hipMalloc(&col, cuda_csr->get_num_nnz() * sizeof(IDType));
  hipMemcpy(col, cuda_csr->get_col(), cuda_csr->get_num_nnz() * sizeof(IDType),
             hipMemcpyDeviceToDevice);
  if (cuda_csr->get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      hipMalloc(&vals, cuda_csr->get_num_nnz() * sizeof(ValueType));
      hipMemcpy(vals, cuda_csr->get_vals(),
                 cuda_csr->get_num_nnz() * sizeof(ValueType),
                 hipMemcpyDeviceToDevice);
    }
  }
  return new format::cuda::CUDACSR<IDType, NNZType, ValueType>(
      cuda_csr->get_dimensions()[0], cuda_csr->get_dimensions()[0],
      cuda_csr->get_num_nnz(), row_ptr, col, vals, *dest_gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
format::Format *CUDACsrCsrConditionalFunction(format::Format *source,
                                      context::Context *context) {
  context::cuda::HIPContext *gpu_context =
      static_cast<context::cuda::HIPContext *>(source->get_context());
  auto cuda_csr =
      source->AsAbsolute<format::cuda::CUDACSR<IDType, NNZType, ValueType>>();
  hipSetDevice(gpu_context->device_id);
  int n = cuda_csr->get_dimensions()[0];
  int nnz = cuda_csr->get_num_nnz();
  NNZType *row_ptr = new NNZType[n + 1];
  IDType *col = new IDType[nnz];
  ValueType *vals = nullptr;
  hipMemcpy(row_ptr, cuda_csr->get_row_ptr(), (n + 1) * sizeof(NNZType),
             hipMemcpyDeviceToHost);
  hipMemcpy(col, cuda_csr->get_col(), nnz * sizeof(IDType),
             hipMemcpyDeviceToHost);
  if (cuda_csr->get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      vals = new ValueType[nnz];
      hipMemcpy(vals, cuda_csr->get_vals(), nnz * sizeof(ValueType),
                 hipMemcpyDeviceToHost);
    }
  }
  return new format::CSR<IDType, NNZType, ValueType>(n, n, row_ptr, col, vals);
}

bool CUDAPeerToPeer(context::Context *from, context::Context *to) {
  if (!(to->get_context_type_member() ==
            context::cuda::HIPContext::get_context_type() ||
        from->get_context_type_member() ==
            context::cuda::HIPContext::get_context_type()))
    return false;
  auto from_gpu = static_cast<context::cuda::HIPContext *>(from);
  auto to_gpu = static_cast<context::cuda::HIPContext *>(to);
  int can_access;
  hipDeviceCanAccessPeer(&can_access, from_gpu->device_id, to_gpu->device_id);
  return can_access;
}
#if !defined(_HEADER_ONLY)
#include "init/cuda/converter.inc"
#endif
}  // namespace cuda
}  // namespace converter
}  // namespace utils
}  // namespace sparsebase
