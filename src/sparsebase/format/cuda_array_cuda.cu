#include "sparsebase/format/cuda_array_cuda.cuh"
#include "sparsebase/utils/utils_cuda.cuh"

namespace sparsebase::format {

template <typename ValueType>
CUDAArray<ValueType>::CUDAArray(CUDAArray<ValueType> &&rhs)
    : vals_(std::move(rhs.vals_)) {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  this->nnz_ = rhs.get_num_nnz();
  this->order_ = 1;
  this->dimension_ = rhs.dimension_;
  rhs.vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      nullptr, BlankDeleter<ValueType>());
  this->context_ = std::unique_ptr<sparsebase::context::Context>(
      new sparsebase::context::CPUContext);
}
template <typename ValueType>
CUDAArray<ValueType> &CUDAArray<ValueType>::operator=(
    const CUDAArray<ValueType> &rhs) {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  this->nnz_ = rhs.nnz_;
  this->order_ = 1;
  this->dimension_ = rhs.dimension_;
  ValueType *vals = nullptr;
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(this->get_context());
  if (rhs.get_vals() != nullptr) {
    hipSetDevice(gpu_context->device_id);
    hipMalloc(&vals, rhs.get_num_nnz() * sizeof(ValueType));
    hipMemcpy(vals, rhs.get_vals(), rhs.get_num_nnz() * sizeof(ValueType),
               hipMemcpyDeviceToDevice);
    vals = new ValueType[rhs.get_num_nnz()];
    std::copy(rhs.get_vals(), rhs.get_vals() + rhs.get_num_nnz(), vals);
  }
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, utils::CUDADeleter<ValueType>());
  return *this;
}
template <typename ValueType>
CUDAArray<ValueType>::CUDAArray(const CUDAArray<ValueType> &rhs)
    : vals_(nullptr, BlankDeleter<ValueType>()) {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  this->nnz_ = rhs.nnz_;
  this->order_ = 1;
  this->dimension_ = rhs.dimension_;
  ValueType *vals = nullptr;
  if (rhs.get_vals() != nullptr) {
    vals = new ValueType[rhs.get_num_nnz()];
    std::copy(rhs.get_vals(), rhs.get_vals() + rhs.get_num_nnz(), vals);
  }
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, Deleter<ValueType>());
  this->context_ = std::unique_ptr<sparsebase::context::Context>(
      new sparsebase::context::CPUContext);
}
template <typename ValueType>
CUDAArray<ValueType>::CUDAArray(DimensionType nnz, ValueType *vals,
                                context::HIPContext context, Ownership own)
    : vals_(vals, BlankDeleter<ValueType>()) {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  this->order_ = 1;
  this->dimension_ = {(DimensionType)nnz};
  this->nnz_ = nnz;
  if (own == kOwned) {
    this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
        vals, utils::CUDADeleter<ValueType>());
  }
  this->context_ = std::unique_ptr<sparsebase::context::Context>(
      new sparsebase::context::HIPContext(context));
}

template <typename ValueType>
Format *CUDAArray<ValueType>::Clone() const {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  return new CUDAArray(*this);
}
template <typename ValueType>
ValueType *CUDAArray<ValueType>::get_vals() const {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  return vals_.get();
}
template <typename ValueType>
ValueType *CUDAArray<ValueType>::release_vals() {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  auto vals = vals_.release();
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, BlankDeleter<ValueType>());
  return vals;
}

template <typename ValueType>
void CUDAArray<ValueType>::set_vals(ValueType *vals, Ownership own) {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  if (own == kOwned) {
    this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
        vals, Deleter<ValueType>());
  } else {
    this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
        vals, BlankDeleter<ValueType>());
  }
}

template <typename ValueType>
bool CUDAArray<ValueType>::ValsIsOwned() {
  static_assert(!std::is_same_v<ValueType, void>,
                "Cannot create CUDAArray with void ValueType");
  return (this->vals_.get_deleter().target_type() !=
          typeid(BlankDeleter<ValueType>));
}
template <typename ValueType>
CUDAArray<ValueType>::~CUDAArray() {}
// format.inc
#ifndef _HEADER_ONLY
#include "init/cuda/cuda_array_cuda.inc"
#endif
}  // namespace sparsebase::format