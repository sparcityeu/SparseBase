#include <iostream>

#include "sparsebase/format/cuda_csr_cuda.cuh"
#include "sparsebase/format/format.h"
#include "sparsebase/format/format_order_one.h"
#include "sparsebase/format/format_order_two.h"
#include "sparsebase/utils/exception.h"
#include "sparsebase/utils/utils_cuda.cuh"

namespace sparsebase::format {

template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::CUDACSR(
    CUDACSR<IDType, NNZType, ValueType> &&rhs)
    : col_(std::move(rhs.col_)),
      row_ptr_(std::move(rhs.row_ptr_)),
      vals_(std::move(rhs.vals_)) {
  this->nnz_ = rhs.get_num_nnz();
  this->order_ = 2;
  this->dimension_ = rhs.dimension_;
  rhs.col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      nullptr, BlankDeleter<IDType>());
  rhs.row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      nullptr, BlankDeleter<NNZType>());
  rhs.vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      nullptr, BlankDeleter<ValueType>());
  this->context_ = std::unique_ptr<sparsebase::context::Context>(
      new sparsebase::context::HIPContext(
          rhs.get_cuda_context()->device_id));
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>
    &CUDACSR<IDType, NNZType, ValueType>::operator=(
        const CUDACSR<IDType, NNZType, ValueType> &rhs) {
  this->nnz_ = rhs.nnz_;
  this->order_ = 2;
  this->dimension_ = rhs.dimension_;
  IDType *col;
  NNZType *row_ptr;
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(this->get_cuda_context());
  hipSetDevice(gpu_context->device_id);
  hipMalloc(&col, rhs.get_num_nnz() * sizeof(IDType));
  hipMemcpy(col, rhs.get_col(), rhs.get_num_nnz() * sizeof(IDType),
             hipMemcpyDeviceToDevice);
  hipMalloc(&row_ptr, (rhs.get_dimensions()[0] + 1) * sizeof(NNZType));
  hipMemcpy(row_ptr, rhs.get_row_ptr(),
             (rhs.get_dimensions()[0] + 1) * sizeof(NNZType),
             hipMemcpyDeviceToDevice);
  ValueType *vals = nullptr;
  if (rhs.get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      hipMalloc(&vals, rhs.get_num_nnz() * sizeof(ValueType));
      hipMemcpy(vals, rhs.get_vals(), rhs.get_num_nnz() * sizeof(ValueType),
                 hipMemcpyDeviceToDevice);
    }
  }
  this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      col, utils::CUDADeleter<IDType>());
  this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      row_ptr, utils::CUDADeleter<NNZType>());
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, utils::CUDADeleter<ValueType>());
  return *this;
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::CUDACSR(
    const CUDACSR<IDType, NNZType, ValueType> &rhs)
    : col_(nullptr, BlankDeleter<IDType>()),
      row_ptr_(nullptr, BlankDeleter<NNZType>()),
      vals_(nullptr, BlankDeleter<ValueType>()) {
  this->nnz_ = rhs.nnz_;
  this->order_ = 2;
  this->dimension_ = rhs.dimension_;
  IDType *col;
  NNZType *row_ptr;
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(this->get_context());
  hipSetDevice(gpu_context->device_id);
  hipMalloc(&col, rhs.get_num_nnz() * sizeof(IDType));
  hipMemcpy(col, rhs.get_col(), rhs.get_num_nnz() * sizeof(IDType),
             hipMemcpyDeviceToDevice);
  hipMalloc(&row_ptr, rhs.get_dimensions()[0] * sizeof(NNZType));
  hipMemcpy(row_ptr, rhs.get_row_ptr(),
             (rhs.get_dimensions()[0] + 1) * sizeof(NNZType),
             hipMemcpyDeviceToDevice);
  ValueType *vals = nullptr;
  if (rhs.get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      hipMalloc(&vals, rhs.get_num_nnz() * sizeof(ValueType));
      hipMemcpy(vals, rhs.get_vals(), rhs.get_num_nnz() * sizeof(ValueType),
                 hipMemcpyDeviceToDevice);
    }
  }
  this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      col, utils::CUDADeleter<IDType>());
  this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      row_ptr, utils::CUDADeleter<NNZType>());
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, utils::CUDADeleter<ValueType>());
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::CUDACSR(IDType n, IDType m, NNZType nnz,
                                             NNZType *row_ptr, IDType *col,
                                             ValueType *vals,
                                             context::HIPContext context,
                                             Ownership own)
    : row_ptr_(row_ptr, BlankDeleter<NNZType>()),
      col_(col, BlankDeleter<IDType>()),
      vals_(vals, BlankDeleter<ValueType>()) {
  this->order_ = 2;
  this->dimension_ = {(DimensionType)n, (DimensionType)m};
  this->nnz_ = nnz;
  if (own == kOwned) {
    this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
        row_ptr, utils::CUDADeleter<NNZType>());
    this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
        col, utils::CUDADeleter<IDType>());
    this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
        vals, utils::CUDADeleter<ValueType>());
  }
  this->context_ = std::unique_ptr<sparsebase::context::Context>(
      new sparsebase::context::HIPContext(context));
}

template <typename IDType, typename NNZType, typename ValueType>
context::HIPContext *
CUDACSR<IDType, NNZType, ValueType>::get_cuda_context() const {
  return static_cast<context::HIPContext *>(this->get_context());
}
template <typename IDType, typename NNZType, typename ValueType>
Format *CUDACSR<IDType, NNZType, ValueType>::Clone() const {
  return new CUDACSR(*this);
}
template <typename IDType, typename NNZType, typename ValueType>
IDType *CUDACSR<IDType, NNZType, ValueType>::get_col() const {
  return col_.get();
}
template <typename IDType, typename NNZType, typename ValueType>
NNZType *CUDACSR<IDType, NNZType, ValueType>::get_row_ptr() const {
  return row_ptr_.get();
}
template <typename IDType, typename NNZType, typename ValueType>
ValueType *CUDACSR<IDType, NNZType, ValueType>::get_vals() const {
  return vals_.get();
}
template <typename IDType, typename NNZType, typename ValueType>
IDType *CUDACSR<IDType, NNZType, ValueType>::release_col() {
  auto col = col_.release();
  this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
      col, BlankDeleter<IDType>());
  return col;
}
template <typename IDType, typename NNZType, typename ValueType>
NNZType *CUDACSR<IDType, NNZType, ValueType>::release_row_ptr() {
  auto row_ptr = row_ptr_.release();
  this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
      row_ptr, BlankDeleter<NNZType>());
  return row_ptr;
}
template <typename IDType, typename NNZType, typename ValueType>
ValueType *CUDACSR<IDType, NNZType, ValueType>::release_vals() {
  auto vals = vals_.release();
  this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
      vals, BlankDeleter<ValueType>());
  return vals;
}

template <typename IDType, typename NNZType, typename ValueType>
void CUDACSR<IDType, NNZType, ValueType>::set_col(
    IDType *col, context::HIPContext context, Ownership own) {
  if (own == kOwned) {
    this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
        col, utils::CUDADeleter<IDType>());
  } else {
    this->col_ = std::unique_ptr<IDType, std::function<void(IDType *)>>(
        col, BlankDeleter<IDType>());
  }
}

template <typename IDType, typename NNZType, typename ValueType>
void CUDACSR<IDType, NNZType, ValueType>::set_row_ptr(
    NNZType *row_ptr, context::HIPContext context, Ownership own) {
  if (own == kOwned) {
    this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
        row_ptr, utils::CUDADeleter<NNZType>());
  } else {
    this->row_ptr_ = std::unique_ptr<NNZType, std::function<void(NNZType *)>>(
        row_ptr, BlankDeleter<NNZType>());
  }
}

template <typename IDType, typename NNZType, typename ValueType>
void CUDACSR<IDType, NNZType, ValueType>::set_vals(
    ValueType *vals, context::HIPContext context, Ownership own) {
  if (own == kOwned) {
    this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
        vals, utils::CUDADeleter<ValueType>());
  } else {
    this->vals_ = std::unique_ptr<ValueType, std::function<void(ValueType *)>>(
        vals, BlankDeleter<ValueType>());
  }
}

template <typename IDType, typename NNZType, typename ValueType>
bool CUDACSR<IDType, NNZType, ValueType>::RowPtrIsOwned() {
  return (this->row_ptr_.get_deleter().target_type() !=
          typeid(BlankDeleter<NNZType>));
}

template <typename IDType, typename NNZType, typename ValueType>
bool CUDACSR<IDType, NNZType, ValueType>::ColIsOwned() {
  return (this->col_.get_deleter().target_type() !=
          typeid(BlankDeleter<IDType>));
}

template <typename IDType, typename NNZType, typename ValueType>
bool CUDACSR<IDType, NNZType, ValueType>::ValsIsOwned() {
  return (this->vals_.get_deleter().target_type() !=
          typeid(BlankDeleter<ValueType>));
}
template <typename IDType, typename NNZType, typename ValueType>
CUDACSR<IDType, NNZType, ValueType>::~CUDACSR() {}

#ifndef _HEADER_ONLY
#include "init/cuda/cuda_csr_cuda.inc"
#endif
}  // namespace cuda