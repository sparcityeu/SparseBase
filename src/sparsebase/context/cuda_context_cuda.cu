//
// Created by Amro on 3/31/2022.
//
#include "cuda_context_cuda.cuh"
#include "sparsebase/context/context.h"
#include "sparsebase/utils/exception.h"
namespace sparsebase::context {

HIPContext::HIPContext(int did) : device_id(did) {
  int device_count;
  hipGetDeviceCount(&device_count);
  if (device_id >= device_count) {
    throw utils::CUDADeviceException(device_count, device_id);
  }
}
bool HIPContext::IsEquivalent(Context *rhs) const {
  if (dynamic_cast<HIPContext *>(rhs) != nullptr) {
    if (dynamic_cast<HIPContext *>(rhs)->device_id == this->device_id)
      return true;
  }
  return false;
}
}  // namespace sparsebase::context