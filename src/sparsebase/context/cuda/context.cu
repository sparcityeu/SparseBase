//
// Created by Amro on 3/31/2022.
//
#include "sparsebase/context/context.h"
#include "sparsebase/context/cuda/context.cuh"
namespace sparsebase {
namespace context {
namespace cuda {

HIPContext::HIPContext(int did) : device_id(did) {
  int device_count;
  hipGetDeviceCount(&device_count);
  if (device_id >= device_count) {
    throw utils::CUDADeviceException(device_count, device_id);
  }
}
bool HIPContext::IsEquivalent(Context *rhs) const {
  if (dynamic_cast<HIPContext *>(rhs) != nullptr) {
    if (dynamic_cast<HIPContext *>(rhs)->device_id == this->device_id)
      return true;
  }
  return false;
}
} // namespace cuda
} // namespace context
} // namespace sparsebase