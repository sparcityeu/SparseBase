#include "sparsebase/converter/converter.h"
#include "sparsebase/converter/converter_order_one_cuda.cuh"
#include "sparsebase/format/array.h"
#include "sparsebase/format/cuda_array_cuda.cuh"
#include "sparsebase/format/format.h"
#include "sparsebase/format/format_order_one.h"

namespace sparsebase::converter {
template <typename ValueType>
format::Format *CUDAArrayArrayConditionalFunction(format::Format *source,
                                                  context::Context *context) {
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(source->get_context());
  auto cuda_array = source->AsAbsolute<format::CUDAArray<ValueType>>();
  hipSetDevice(gpu_context->device_id);
  ValueType *vals = nullptr;
  if (cuda_array->get_vals() != nullptr) {
    vals = new ValueType[cuda_array->get_num_nnz()];
    hipMemcpy(vals, cuda_array->get_vals(),
               cuda_array->get_num_nnz() * sizeof(ValueType),
               hipMemcpyDeviceToHost);
  }
  return new format::Array<ValueType>(cuda_array->get_num_nnz(), vals,
                                      format::kOwned);
}
template <typename ValueType>
format::Format *ArrayCUDAArrayConditionalFunction(format::Format *source,
                                                  context::Context *context) {
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(context);
  auto array = source->AsAbsolute<format::Array<ValueType>>();
  hipSetDevice(gpu_context->device_id);
  ValueType *vals = nullptr;
  if (array->get_vals() != nullptr) {
    hipMalloc(&vals, array->get_num_nnz() * sizeof(ValueType));
    hipMemcpy(vals, array->get_vals(),
               array->get_num_nnz() * sizeof(ValueType),
               hipMemcpyHostToDevice);
  }
  return new format::CUDAArray<ValueType>(array->get_num_nnz(), vals,
                                          *gpu_context, format::kOwned);
}
#if !defined(_HEADER_ONLY)
#include "init/cuda/converter_order_one_cuda.inc"
#endif
}  // namespace sparsebase::converter