#include "sparsebase/converter/converter.h"
#include "sparsebase/converter/converter_order_two.h"
#include "sparsebase/converter/converter_order_two_cuda.cuh"
#include "sparsebase/format/cuda_csr_cuda.cuh"
#include "sparsebase/format/format.h"
#include "sparsebase/format/format_order_one.h"
#include "sparsebase/format/format_order_two.h"

namespace sparsebase::converter {
template <typename IDType, typename NNZType, typename ValueType>
format::Format *CsrCUDACsrConditionalFunction(format::Format *source,
                                      context::Context *context) {
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(context);
  auto csr = source->AsAbsolute<format::CSR<IDType, NNZType, ValueType>>();
  hipSetDevice(gpu_context->device_id);
  NNZType *row_ptr;
  IDType *col;
  ValueType *vals = nullptr;
  hipMalloc(&row_ptr, (1 + csr->get_dimensions()[0]) * sizeof(NNZType));
  hipMemcpy(row_ptr, csr->get_row_ptr(),
             (1 + csr->get_dimensions()[0]) * sizeof(NNZType),
             hipMemcpyHostToDevice);
  hipMalloc(&col, csr->get_num_nnz() * sizeof(IDType));
  hipMemcpy(col, csr->get_col(), csr->get_num_nnz() * sizeof(IDType),
             hipMemcpyHostToDevice);
  if (csr->get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      hipMalloc(&vals, csr->get_num_nnz() * sizeof(ValueType));
      hipMemcpy(vals, csr->get_vals(), csr->get_num_nnz() * sizeof(ValueType),
                 hipMemcpyHostToDevice);
    }
  }
  return new format::CUDACSR<IDType, NNZType, ValueType>(
      csr->get_dimensions()[0], csr->get_dimensions()[0], csr->get_num_nnz(),
      row_ptr, col, vals, *gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
format::Format *CUDACsrCUDACsrConditionalFunction(format::Format *source,
                                          context::Context *context) {
  context::HIPContext *dest_gpu_context =
      static_cast<context::HIPContext *>(context);
  auto cuda_csr =
      source->AsAbsolute<format::CUDACSR<IDType, NNZType, ValueType>>();
  context::HIPContext *source_gpu_context =
      static_cast<context::HIPContext *>(cuda_csr->get_context());
  hipSetDevice(dest_gpu_context->device_id);
  hipDeviceEnablePeerAccess(source_gpu_context->device_id, 0);
  NNZType *row_ptr;
  IDType *col;
  ValueType *vals = nullptr;
  hipMalloc(&row_ptr, cuda_csr->get_dimensions()[0] * sizeof(NNZType));
  hipMemcpy(row_ptr, cuda_csr->get_row_ptr(),
             (1 + cuda_csr->get_dimensions()[0]) * sizeof(NNZType),
             hipMemcpyDeviceToDevice);
  hipMalloc(&col, cuda_csr->get_num_nnz() * sizeof(IDType));
  hipMemcpy(col, cuda_csr->get_col(), cuda_csr->get_num_nnz() * sizeof(IDType),
             hipMemcpyDeviceToDevice);
  if (cuda_csr->get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      hipMalloc(&vals, cuda_csr->get_num_nnz() * sizeof(ValueType));
      hipMemcpy(vals, cuda_csr->get_vals(),
                 cuda_csr->get_num_nnz() * sizeof(ValueType),
                 hipMemcpyDeviceToDevice);
    }
  }
  return new format::CUDACSR<IDType, NNZType, ValueType>(
      cuda_csr->get_dimensions()[0], cuda_csr->get_dimensions()[0],
      cuda_csr->get_num_nnz(), row_ptr, col, vals, *dest_gpu_context);
}
template <typename IDType, typename NNZType, typename ValueType>
format::Format *CUDACsrCsrConditionalFunction(format::Format *source,
                                      context::Context *context) {
  context::HIPContext *gpu_context =
      static_cast<context::HIPContext *>(source->get_context());
  auto cuda_csr =
      source->AsAbsolute<format::CUDACSR<IDType, NNZType, ValueType>>();
  hipSetDevice(gpu_context->device_id);
  int n = cuda_csr->get_dimensions()[0];
  int nnz = cuda_csr->get_num_nnz();
  NNZType *row_ptr = new NNZType[n + 1];
  IDType *col = new IDType[nnz];
  ValueType *vals = nullptr;
  hipMemcpy(row_ptr, cuda_csr->get_row_ptr(), (n + 1) * sizeof(NNZType),
             hipMemcpyDeviceToHost);
  hipMemcpy(col, cuda_csr->get_col(), nnz * sizeof(IDType),
             hipMemcpyDeviceToHost);
  if (cuda_csr->get_vals() != nullptr) {
    if constexpr (std::is_same_v<ValueType, void>) {
      throw utils::TypeException("Cannot create values array for type void");
    } else {
      vals = new ValueType[nnz];
      hipMemcpy(vals, cuda_csr->get_vals(), nnz * sizeof(ValueType),
                 hipMemcpyDeviceToHost);
    }
  }
  return new format::CSR<IDType, NNZType, ValueType>(n, n, row_ptr, col, vals);
}
#if !defined(_HEADER_ONLY)
#include "init/cuda/converter_order_two_cuda.inc"
#endif
}