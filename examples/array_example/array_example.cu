#include "hip/hip_runtime.h"
#include <iostream>

#include "sparsebase/sparse_format.h"
#include "sparsebase/sparse_converter.h"
#include "sparsebase/cuda.cuh"

using namespace std;
using namespace sparsebase;

void print_array(int * vals, int n){
    printf("Printing the Array on the CPU\n");
    for (int i = 0; i < n; i++){
        printf("%d ", vals[i]);
    }
    printf("\n");
}
__global__ void print_array_cuda(int * vals, int n){
    printf("Printing the Array on the GPU\n");
    for (int i = 0; i < n; i++){
        printf("%d ", vals[i]);
    }
    printf("\n");
}

int main(){

    int vals[6] = {10, 20, 30, 40, 50, 60};
    context::HIPContext gpu_context{0};
    context::CPUContext cpu_context;

    format::Array<int>* array = new format::Array<int>(6, vals);

    auto converter = new utils::Converter<int,int,int>();

    auto cuda_array = converter->ConvertConditional<format::CUDAArray<int>>(array, &gpu_context);

    print_array_cuda<<<1,1>>>(cuda_array->get_vals(), cuda_array->get_dimensions()[0]);
    hipDeviceSynchronize();

    auto cpu_array = converter->ConvertConditional<format::Array<int>>(cuda_array, &cpu_context);

    print_array(cpu_array->get_vals(), cuda_array->get_dimensions()[0]);

    return 0;
}

