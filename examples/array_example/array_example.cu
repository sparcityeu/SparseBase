#include "hip/hip_runtime.h"
#include <iostream>

#include "sparsebase/format/cuda/format.cuh"
#include "sparsebase/format/format.h"
#include "sparsebase/utils/converter/converter.h"

using namespace std;
using namespace sparsebase;

void print_array(int *vals, int n) {
  printf("Printing the Array on the CPU\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", vals[i]);
  }
  printf("\n");
}
__global__ void print_array_cuda(int *vals, int n) {
  printf("Printing the Array on the GPU\n");
  for (int i = 0; i < n; i++) {
    printf("%d ", vals[i]);
  }
  printf("\n");
}

int main() {

  int vals[6] = {10, 20, 30, 40, 50, 60};
  context::cuda::HIPContext gpu_context{0};
  context::CPUContext cpu_context;

  format::Array<int> *array = new format::Array<int>(6, vals);

  auto converter = new utils::converter::ConverterOrderOne<int>();

  auto cuda_array =
      converter->Convert<format::cuda::CUDAArray<int>>(array, &gpu_context);

  print_array_cuda<<<1, 1>>>(cuda_array->get_vals(),
                             cuda_array->get_dimensions()[0]);
  hipDeviceSynchronize();

  auto cpu_array =
      converter->Convert<format::Array<int>>(cuda_array, &cpu_context);

  print_array(cpu_array->get_vals(), cuda_array->get_dimensions()[0]);

  return 0;
}
