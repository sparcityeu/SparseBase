#include "hip/hip_runtime.h"
#include <iostream>

#include "sparsebase/format/format.h"
#include "sparsebase/utils/converter/converter.h"
#include "sparsebase/preprocess/preprocess.h"
#include "sparsebase/cuda/format/format.cuh"

using namespace std;
using namespace sparsebase;

template <typename T>
void print_array(T * vals, int n){
    printf("Printing the Array on the CPU\n");
    for (int i = 0; i < n; i++){
        printf("%f ", vals[i]);
    }
    printf("\n");
}
template <typename T>
__global__ void print_array_cuda(T * vals, int n){
    printf("Printing the Array on the GPU\n");
    for (int i = 0; i < n; i++){
        printf("%f ", vals[i]);
    }
    printf("\n");
}
void print_csr(int * row_ptr, int * col, int n){
    printf("Printing the CSR on the CPU\n");
    for (int i = 0; i < n; i++){
        printf("%d: ", i);
        for (int j = row_ptr[i]; j < row_ptr[i+1];j++){
            printf("%d ", col[j]);
        }
        printf("\n");
    }
}
__global__ void print_csr_cuda(int * row_ptr, int * col, int n){
    printf("Printing the CSR on the GPU\n");
    for (int i = 0; i < n; i++){
        printf("%d: ", i);
        for (int j = row_ptr[i]; j < row_ptr[i+1];j++){
            printf("%d ", col[j]);
        }
        printf("\n");
    }
}

int main(){

    int row_ptr[6] = {0, 2, 4, 6, 6, 6};
    int col[6] = {1, 3, 0, 2, 0, 1};
    int vals[6] = {10, 20, 30, 40, 50, 60};
    context::HIPContext gpu_context{0};
    context::CPUContext cpu_context;

    format::CSR<int,int,int>* csr = new format::CSR<int,int,int>(5, 5, row_ptr, col, vals);

    auto graph_converter = new utils::OrderTwoConverter<int,int,int>();
    auto array_converter = new utils::OrderOneConverter<float>();


    preprocess::JaccardWeights<int, int, int, float> jac;
    auto array = jac.GetJaccardWeights({csr}, {&gpu_context, &cpu_context});

    if (array->get_context_type() == context::CPUContext::get_context_type()){
        auto cpu_array = array_converter->Convert<format::Array<float>>(array, &cpu_context);
        print_array(cpu_array->get_vals(), cpu_array->get_num_nnz());
    }
    if (array->get_context_type() == context::HIPContext::get_context_type()){
        auto gpu_array = array_converter->Convert<format::CUDAArray<float>>(array, &gpu_context);
        print_array_cuda<<<1,1>>>(gpu_array->get_vals(), gpu_array->get_num_nnz());
    }


    auto cuda_csr = graph_converter->Convert<format::CUDACSR<int, int, int>>(csr, &gpu_context);

    print_csr_cuda<<<1,1>>>(cuda_csr->get_row_ptr(), cuda_csr->get_col(), cuda_csr->get_dimensions()[0]);
    hipDeviceSynchronize();

    auto cpu_csr = graph_converter->Convert<format::CSR<int, int, int>>(cuda_csr, &cpu_context);

    print_csr(cpu_csr->get_row_ptr(), cpu_csr->get_col(), cuda_csr->get_dimensions()[0]);

    return 0;
}

