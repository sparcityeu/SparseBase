#include "hip/hip_runtime.h"
#include <iostream>

#include "sparsebase/sparse_format.h"
#include "sparsebase/sparse_converter.h"
#include "sparsebase/sparse_preprocess.h"
#include "sparsebase/cuda.cuh"

using namespace std;
using namespace sparsebase;

template <typename T>
void print_array(T * vals, int n){
    printf("Printing the Array on the CPU\n");
    for (int i = 0; i < n; i++){
        printf("%f ", vals[i]);
    }
    printf("\n");
}
template <typename T>
__global__ void print_array_cuda(T * vals, int n){
    printf("Printing the Array on the GPU\n");
    for (int i = 0; i < n; i++){
        printf("%f ", vals[i]);
    }
    printf("\n");
}
void print_csr(int * row_ptr, int * col, int n){
    printf("Printing the CSR on the CPU\n");
    for (int i = 0; i < n; i++){
        printf("%d: ", i);
        for (int j = row_ptr[i]; j < row_ptr[i+1];j++){
            printf("%d ", col[j]);
        }
        printf("\n");
    }
}
__global__ void print_csr_cuda(int * row_ptr, int * col, int n){
    printf("Printing the CSR on the GPU\n");
    for (int i = 0; i < n; i++){
        printf("%d: ", i);
        for (int j = row_ptr[i]; j < row_ptr[i+1];j++){
            printf("%d ", col[j]);
        }
        printf("\n");
    }
}

int main(){

    int row_ptr[6] = {0, 2, 4, 6, 6, 6};
    int col[6] = {1, 3, 0, 2, 0, 1};
    int vals[6] = {10, 20, 30, 40, 50, 60};
    context::HIPContext gpu_context{0};
    context::CPUContext cpu_context;

    format::CSR<int,int,int>* csr = new format::CSR<int,int,int>(5, 5, row_ptr, col, vals);

    auto converter = new utils::Converter<int,int,int>();
    auto converter2 = new utils::Converter<int,int,float>();


    preprocess::JaccardWeights<int, int, int, float> jac;
    auto cuda_array = jac.GetJaccardWeights({csr}, {&gpu_context, &cpu_context});
    auto cpu_array = converter2->ConvertConditional<format::Array<float>>(cuda_array, &cpu_context);

    print_array(cpu_array->get_vals(), cpu_array->get_num_nnz());

    auto cuda_csr = converter->ConvertConditional<format::CUDACSR<int, int, int>>(csr, &gpu_context);

    print_csr_cuda<<<1,1>>>(cuda_csr->get_row_ptr(), cuda_csr->get_col(), cuda_csr->get_dimensions()[0]);
    hipDeviceSynchronize();

    auto cpu_csr = converter->ConvertConditional<format::CSR<int, int, int>>(cuda_csr, &cpu_context);

    print_csr(cpu_csr->get_row_ptr(), cpu_csr->get_col(), cuda_csr->get_dimensions()[0]);

    return 0;
}

